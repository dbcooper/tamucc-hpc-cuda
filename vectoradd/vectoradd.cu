
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int* a, int* b, int* c)
{
        *c = *a + *b;
}

int main(void)
{
        int a, b, c;                    // host copies of data
        int *d_a, *d_b, *d_c;           // devices coipes of data
        int size = sizeof(int);

        printf("Integer size is %d bytes\n", size);

        // Allocate space for device copies of data
        hipMalloc((void**) &d_a, size);
        hipMalloc((void**) &d_b, size);
        hipMalloc((void**) &d_c, size);

        // Setup integers
        a = 2;
        b = 7;

        printf("a=%d;  b=%d\n", a, b);

        // Copy inputs to device
        hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

        // Launch add() on device
        add<<<1,1>>>(d_a, d_b, d_c);

        // Copy result to host
        hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

        // Cleanup
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);

        // Print out results
        printf("a + b = c\n");
        printf("%d + %d = %d\n", a, b, c);

        return 0;
}
