
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int* a, int* b, int* c)
{
        c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

// Put count random integers into integer [array] pointed to by p
void random_ints(int *p, int count)
{
        for (int i=0; i < count; i++) {
                p[i] = rand() % 32768;  // limit random numbers to [0, 32768]
        }
}

#define N 512

int main(void)
{
        int *a, *b, *c;                 // host copies of data
        int *d_a, *d_b, *d_c;           // devices copies of data
        int size = N * sizeof(int);     // Number of bytes for N integers

        printf("Total integer space size is %d bytes\n", size);

        // Allocate space for device copies of data
        hipMalloc((void**) &d_a, size);
        hipMalloc((void**) &d_b, size);
        hipMalloc((void**) &d_c, size);

        // Allocate space for host copies of input values (a, b, c arrays)
        a = (int *)malloc(size);
        b = (int *)malloc(size);
        c = (int *)malloc(size);

        // Setup integers
        random_ints(a, N);
        random_ints(b, N);

        // Copy inputs to device
        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

        // Launch add() on device
        add<<<N,1>>>(d_a, d_b, d_c);

        // Copy result to host
        hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

        // Print out results
        printf("\nResults of parallel GPU addition across %d elements:\n", N);
        for (int i=0; i < N; i++) {
                int sum = a[i] + b[i];
                if (c[i] == sum) {
                        printf("%4d: %6d + %6d = %6d\n", i, a[i], b[i], c[i]);
                }
                else {
                        printf("%4d: %6d + %6d != %6d  ERROR, should be %d", i, a[i], b[i], c[i], sum);
                }
        }

        // Cleanup
        free(a);  free(b);  free(c);
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);

        return 0;
}
