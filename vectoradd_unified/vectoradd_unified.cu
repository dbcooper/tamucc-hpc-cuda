
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void VecAdd(int* ret, int a, int b)
{
        ret[threadIdx.x] = a + b + threadIdx.x;
}

int main(void)
{
        int  a   = 10;
        int  b   = 100;
        int* ret = NULL;                // results of addition

        hipMallocManaged(&ret, 1000 * sizeof(int));
        VecAdd<<< 1, 1000 >>>(ret, a, b);
        hipDeviceSynchronize();
        for (int i = 0; i < 1000; i++) {
                printf("%4d: %d + %d + %4d = %5d\n", i, a, b, i, ret[i]);
        }
        hipFree(ret);
        return 0;
}
